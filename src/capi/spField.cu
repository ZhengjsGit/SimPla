/*
 * spField.cu
 *
 *  Created on: 2016年6月15日
 *      Author: salmon
 */
#include "sp_def.h"
#include "sp_cuda_common.h"
#include "spMesh.h"
#include "spField.h"

void
spCreateField(const spMesh *ctx, sp_field_type **f, int iform)
{
  CUDA_CHECK_RETURN(hipMalloc (f, sizeof(sp_field_type)));
  CUDA_CHECK_RETURN(
	  hipMalloc (
		  &((*f)->data),
		  ctx->number_of_cell * ((iform == 1 || iform == 2) ? 3 : 1)
			  * sizeof(Real)));
}

void
spDestroyField(const spMesh *ctx, sp_field_type **f)
{
  CUDA_CHECK_RETURN(hipFree ((*f)->data));
  CUDA_CHECK_RETURN(hipFree ((*f)));
  *f = 0x0;
}
