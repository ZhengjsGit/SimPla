/*
 * spMesh.c
 *
 *  Created on: 2016年6月15日
 *      Author: salmon
 */

#include "../sp_config.h"
#include "sp_def_cuda.h"
#include "spMesh.h"
void spCreateMesh(spMesh **ctx)
{
	CUDA_CHECK_RETURN(hipMalloc(ctx, sizeof(spMesh)));
}
void spDestroyMesh(spMesh **ctx)
{
	CUDA_CHECK_RETURN(hipFree(*ctx));
	*ctx = 0x0;
}
