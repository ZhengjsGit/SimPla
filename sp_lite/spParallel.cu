//
// Created by salmon on 16-7-6.
//
#include "sp_lite_def.h"
#include "spParallel.h"

void spParallelInitialize()
{
    CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
    CUDA_CHECK_RETURN(hipGetLastError());
}

void spParallelFinalize()
{
    CUDA_CHECK_RETURN(hipDeviceReset());

}

void spParallelGlobalSync()
{
    CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
}


void spParallelDeviceMalloc(void **p, size_type s)
{
    CUDA_CHECK_RETURN(hipMalloc(p, s));

}

void spParallelDeviceFree(void *p)
{
    CUDA_CHECK_RETURN(hipFree(p));
}

void spParallelMemcpy(void *dest, void const *src, size_type s)
{
    CUDA_CHECK_RETURN(hipMemcpy(dest, src, s, hipMemcpyDefault));


}

void spParallelMemcpyToSymbol(void *dest, void const *src, size_type s)
{
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(dest), src, s));

}

void spParallelMemset(void *dest, byte_type v, size_type s)
{
    CUDA_CHECK_RETURN(hipMemset(dest, v, s));

}