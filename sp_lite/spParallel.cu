#include "hip/hip_runtime.h"
//
// Created by salmon on 16-7-6.
//
#include "sp_lite_def.h"
#include "spParallel.h"

#include <mpi.h>

// CUDA runtime
#include </usr/local/cuda/include/hip/hip_runtime.h>

void spParallelInitialize(int argc, char **argv)
{

    spMPIInitialize(argc, argv);

    int num_of_device = 0;
    CUDA_CHECK_RETURN(hipGetDeviceCount(&num_of_device));
    CUDA_CHECK_RETURN(hipSetDevice(spMPIProcessNum() % num_of_device));
    CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
    CUDA_CHECK_RETURN(hipGetLastError());
}

void spParallelFinalize()
{
    CUDA_CHECK_RETURN(hipDeviceReset());
    spMPIFinialize();

}

void spParallelDeviceSync()
{
    CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
}

void spParallelHostMalloc(void **p, size_type s)
{
    CUDA_CHECK_RETURN(hipHostAlloc(p, s, hipHostMallocDefault););

}

void spParallelHostFree(void **p)
{
    if (*p != NULL)
    {
        hipHostFree(*p);
        *p = NULL;
    }
}

MC_HOST void spParallelDeviceMalloc(void **p, size_type s)
{
    CUDA_CHECK_RETURN(hipMalloc(p, s));
}

MC_HOST void spParallelDeviceFree(void **p)
{
    if (*p != NULL)
    {
        CUDA_CHECK_RETURN(hipFree(*p));
        *p = NULL;
    }
}

MC_HOST void spParallelMemcpy(void *dest, void const *src, size_type s)
{
    CUDA_CHECK_RETURN(hipMemcpy(dest, src, s, hipMemcpyDefault));
}

void spParallelMemcpyToSymbol(void *dest, void const *src, size_type s)
{
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(dest), src, s));

}

void spParallelMemset(void *dest, byte_type v, size_type s)
{
    CUDA_CHECK_RETURN(hipMemset(dest, v, s));
}

MC_HOST_DEVICE inline int sp_is_device_ptr(void const *p)
{
    hipPointerAttribute_t attribute;
    CUDA_CHECK(hipPointerGetAttributes(&attribute, p));
    return (attribute.device == hipMemoryTypeDevice);

}

MC_HOST_DEVICE inline int sp_pointer_type(void const *p)
{
    hipPointerAttribute_t attribute;
    CUDA_CHECK(hipPointerGetAttributes(&attribute, p));
    return (attribute.device);

}

//MC_DEVICE float SP_ATOMIC_ADD(float *v, float d) { return atomicAdd(v, d); }

