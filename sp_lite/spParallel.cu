#include "hip/hip_runtime.h"
//
// Created by salmon on 16-7-6.
//
#include "sp_lite_def.h"
#include "spParallel.h"

void spParallelInitialize()
{
    CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
    CUDA_CHECK_RETURN(hipGetLastError());
}

void spParallelFinalize()
{
    CUDA_CHECK_RETURN(hipDeviceReset());

}

void spParallelThreadSync()
{
    CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
}


void spParallelDeviceMalloc(void **p, size_type s)
{
    CUDA_CHECK_RETURN(hipMalloc(p, s));

}

void spParallelDeviceFree(void *p)
{
    if (p != NULL)
    {
        CUDA_CHECK_RETURN(hipFree(p));
    }
}

void spParallelMemcpy(void *dest, void const *src, size_type s)
{
    CUDA_CHECK_RETURN(hipMemcpy(dest, src, s, hipMemcpyDefault));


}

void spParallelMemcpyToSymbol(void *dest, void const *src, size_type s)
{
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(dest), src, s));

}

void spParallelMemset(void *dest, byte_type v, size_type s)
{
    CUDA_CHECK_RETURN(hipMemset(dest, v, s));
}

MC_HOST_DEVICE inline int sp_is_device_ptr(void const *p)
{
    hipPointerAttribute_t attribute;
    CUDA_CHECK(hipPointerGetAttributes(&attribute, p));
    return (attribute.device == hipMemoryTypeDevice);

}

MC_HOST_DEVICE inline int sp_pointer_type(void const *p)
{
    hipPointerAttribute_t attribute;
    CUDA_CHECK(hipPointerGetAttributes(&attribute, p));
    return (attribute.device);

}

MC_DEVICE void spParallelSyncThreads() { __syncthreads(); }


MC_DEVICE float spAtomicAdd(float *v, float d) { return atomicAdd(v, d); }

MC_DEVICE int spAtomicAdd(int *, int) { return atomicAdd(v, d); }