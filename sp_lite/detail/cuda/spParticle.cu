#include "hip/hip_runtime.h"
//
// Created by salmon on 16-9-6.
//

extern "C"
{
#include "../../spParticle.h"
#include "../../spMesh.h"
#include "../../spParallel.h"
#include "../sp_device.h"

}

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include </usr/local/cuda/include/host_defines.h>
#include </usr/local/cuda/include/>

__global__ void spParticleSortKernel(uint *cellStart,        // output: cell start index
                                     uint *cellEnd,          // output: cell end index
                                     uint *trashStart,
                                     uint *gridParticleHash, // input: sorted grid hashes
                                     uint *gridParticleIndex,// input: sorted particle indices
                                     uint numParticles)
{
    extern __shared__ uint sharedHash[];    // blockSize + 1 elements

    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint number_thread = __umul24(gridDim.x, blockDim.x);
    uint hash;

    if (index < numParticles)     // handle case when no. of particles not multiple of block size
    {
        hash = gridParticleHash[gridParticleIndex[index]];

        // Load hash data into shared memory so that we can look
        // at neighboring particle's hash value without loading
        // two hash values per thread
        sharedHash[threadIdx.x + 1] = hash;

        if (index > 0 && threadIdx.x == 0)
        {
            // first thread in block must load neighbor particle hash
            sharedHash[0] = gridParticleHash[gridParticleIndex[index - 1]];
        }
    }

    __syncthreads();

    if (index < numParticles)
    {
        // If this particle has a different cell index to the previous
        // particle then it must be the first particle in the cell,
        // so store the index of this particle in the cell.
        // As it isn't the first particle, it must also be the cell end of
        // the previous particle's cell

        if (index == 0 || hash != sharedHash[threadIdx.x])
        {
            if (hash != -1) { cellStart[hash] = index; } else { *trashStart = index; }
            if (index > 0) cellEnd[sharedHash[threadIdx.x]] = index;
        }

        if (index == numParticles - 1) { if (hash != -1) { cellEnd[hash] = index + 1; }}

//        // Now use the sorted index to reorder the pos and vel data
//        uint sortedIndex = gridParticleIndex[index];
//        float4 pos = FETCH(oldPos, sortedIndex);       // macro does either global read or texture fetch
//        float4 vel = FETCH(oldVel, sortedIndex);       // see particles_kernel.cuh
//
//        sortedPos[index] = pos;
//        sortedVel[index] = vel;
    }
}

int spParticleSort(spParticle *sp)
{
    spMesh const *m = spMeshAttributeGetMesh((spMeshAttribute const *) sp);

    uint iform = spMeshAttributeGetForm((spMeshAttribute const *) sp);

    int num_of_cell = spMeshGetNumberOfEntities(m, SP_DOMAIN_ALL, iform);

    int numParticles = spParticleGetNumOfParticle(sp);

    uint *hash = (uint *) spParticleGetAttributeData(sp, 0);

    uint *start_pos, *end_pos, *index;

    spParticleGetIndexArray(sp, &start_pos, &end_pos, &index);

    thrust::sort_by_key(thrust::device_ptr<uint>(hash),
                        thrust::device_ptr<uint>(hash + numParticles),
                        thrust::device_ptr<uint>(index));

    uint trashStart = 0;
    int numThreads = 256;
    uint smemSize = sizeof(uint) * (numThreads + 1);
    /*@formatter:off*/
    spParticleSortKernel<<<numParticles / numThreads + 1, numThreads,smemSize>>>(
        start_pos, end_pos, &trashStart, hash, index, numParticles);
    /*@formatter:on*/

    return SP_SUCCESS;
};

int spParticleAutoReorder(spParticle *sp)
{
    return SP_DO_NOTHING;
};

SP_DEVICE_DECLARE_KERNEL (spParticleCooridinateConvert,
                          particle_head *sp,
                          Real3 dx, Real3 min,
                          uint const *start_pos,
                          uint const *end_pos,
                          uint const *sorted_index
)
{

    uint s0 = __umul24(blockIdx.x, gridDim.x) +
        __umul24(blockIdx.y, gridDim.y) +
        __umul24(blockIdx.z, gridDim.z);

    __shared__ Real x0, y0, z0;

    if (threadIdx.x == 0)
    {
        x0 = blockIdx.x * dx.x + min.x;
        y0 = blockIdx.y * dx.y + min.y;
        z0 = blockIdx.z * dx.z + min.z;
    }

        spParallelSyncThreads();

    if (start_pos[s0] + threadIdx.x < end_pos[s0])
    {
        int s = sorted_index[start_pos[s0] + threadIdx.x];
        sp->rx[s] += x0;
        sp->ry[s] += y0;
        sp->rz[s] += z0;
    }
};


int spParticleCooridinateLocalToGlobal(spParticle *sp)
{
    spMesh const *m = spMeshAttributeGetMesh((spMeshAttribute const *) sp);

    uint iform = spMeshAttributeGetForm((spMeshAttribute const *) sp);

    Real dx[3], xmin[3], xmax[3];
    int dims[3];

    spMeshGetDims(m, dims);
    spMeshGetDx(m, dx);
    spMeshGetBox(m, SP_DOMAIN_ALL, xmin, xmax);

    void **p_data;

    SP_CALL(spParticleGetAllAttributeData_device(sp, &p_data));

    uint *start_pos, *end_pos, *index;

    spParticleGetIndexArray(sp, &start_pos, &end_pos, &index);

    uint3 blockDim;
    blockDim.x = SP_NUM_OF_THREADS_PER_BLOCK;
    blockDim.y = 1;
    blockDim.z = 1;


    SP_DEVICE_CALL_KERNEL(spParticleCooridinateConvert, intType2Dim3(dims), blockDim,
                          (particle_head *) (p_data), real2Real3(dx), real2Real3(xmin),
                          start_pos, end_pos, index);

    return SP_SUCCESS;

};


__global__ void
spParticleMemcpyKernel(void *dest,
                       void const *src,
                       const uint *gridParticleIndex,// input: sorted particle indices
                       uint numParticles, uint ele_size_in_byte)
{
    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;


    if (index < numParticles)
    {
        memcpy(dest + index * ele_size_in_byte,
               src + gridParticleIndex[index] * ele_size_in_byte, ele_size_in_byte);
    }
}


__global__ void
spParticleMemcpyUIntKernel(uint *dest,
                           uint const *src,
                           const uint *gridParticleIndex,// input: sorted particle indices
                           int numParticles)
{
    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    if (index < numParticles) { dest[index] = src[gridParticleIndex[index]]; }
}

int spParticleReorder(spParticle *sp)
{
    int numThreads = 256;
    int max_num_particle = spParticleGetMaxNumOfParticle(sp);
    int num_particle = spParticleGetNumOfParticle(sp);

    int size_in_byte = 0;
    void *t_data = NULL;
    for (int i = 0; i < spParticleGetNumberOfAttributes(sp); ++i)
    {
        int ele_size_in_byte = spParticleGetAttributeTypeSizeInByte(sp, i);

        int t_size = size_in_byte * ele_size_in_byte;

        if (size_in_byte != t_size)
        {
            size_in_byte = t_size;
            if (t_data != NULL) { spParallelDeviceFree(&t_data); }
            spParallelDeviceAlloc(&t_data, size_in_byte);
        }

        void *src = spParticleGetAttributeData(sp, i);

        if (ele_size_in_byte == sizeof(uint))
        {
            SP_DEVICE_CALL_KERNEL(spParticleMemcpyUIntKernel, num_particle / numThreads + 1, numThreads,
                                  (uint *) t_data, (uint const *) src, spParticleGetSortedIndex(sp),
                                  num_particle);

        }
        else
        {
            SP_DEVICE_CALL_KERNEL(spParticleMemcpyKernel, num_particle / numThreads + 1, numThreads,
                                  t_data, src, spParticleGetSortedIndex(sp), num_particle, ele_size_in_byte);
        }

        SP_CALL(spParticleSetAttributeData(sp, i, t_data));

        t_data = src;
    }


    SP_CALL(spParallelDeviceFree(&t_data));
    return SP_SUCCESS;

};

int spParticleGetCell(spParticle *sp, uint num, uint *cell_hash, uint *start, uint *end, uint **index)
{
    return SP_SUCCESS;
}