#include "hip/hip_runtime.h"
//
// Created by salmon on 16-7-25.
//

#include </usr/local/cuda/include/hip/hip_runtime_api.h>
#include "../../../../../../../usr/local/cuda/include/"

extern "C" {
#include "spParallelCUDA.h"
#include "../../spMPI.h"
}


int spParallelDeviceInitialize(int argc, char **argv)
{
    int num_of_device = 0;
    SP_DEVICE_CALL(hipGetDeviceCount(&num_of_device));
    SP_DEVICE_CALL(hipSetDevice(spMPIRank() % num_of_device));
    SP_DEVICE_CALL(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
    SP_DEVICE_CALL(hipGetLastError());
//    SP_DEVICE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
    return SP_SUCCESS;
}

int spParallelDeviceFinalize()
{
    SP_DEVICE_CALL(hipDeviceReset());
    return SP_SUCCESS;

}

#define SP_DEFAULT_BLOCKS  128
#define SP_DEFAULT_THREADS 128

int spParallelGridDim()
{
    return SP_DEFAULT_THREADS;
};
int spParallelBlockDim()
{
    return SP_DEFAULT_BLOCKS;
};
int spParallelDeviceAlloc(void **p, size_type s)
{
    SP_DEVICE_CALL(hipMalloc(p, s));
    return SP_SUCCESS;
}

int spParallelDeviceFree(void **_P_)
{
    int error_code = SP_SUCCESS;
    if (*_P_ != NULL)
    {
        error_code = SP_DEVICE_CALL(hipFree(*_P_));
        *_P_ = NULL;
    }
    return error_code;
};

int spParallelMemcpy(void *dest, void const *src, size_type s)
{

    return SP_DEVICE_CALL(hipMemcpy(dest, src, s, hipMemcpyDefault));

}

int spParallelMemcpyToCache(const void *dest, void const *src, size_type s)
{

    return SP_DEVICE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dest), src, s));

}

int spParallelMemset(void *dest, int v, size_type s)
{

    return SP_DEVICE_CALL (hipMemset(dest, v, s));
}

int spParallelDeviceSync()
{

    return SP_CALL(spParallelGlobalBarrier())
        || SP_DEVICE_CALL (hipDeviceSynchronize());
}

int spParallelHostAlloc(void **p, size_type s)
{

    return SP_DEVICE_CALL (hipHostAlloc(p, s, hipHostMallocDefault));

};

int spParallelHostFree(void **p)
{
    int error_code = SP_SUCCESS;
    if (*p != NULL)
    {
        error_code = SP_DEVICE_CALL(hipHostFree(*p));
        *p = NULL;
    }
    return error_code;
}


__global__
void spParallelDeviceFillIntKernel(int *d, int v, size_type max)
{
    for (size_t s = threadIdx.x + blockIdx.x * blockDim.x; s < max; s += gridDim.x * blockDim.x) { d[s] = v; }
};

int spParallelDeviceFillInt(int *d, int v, size_type s)
{
    SP_DEVICE_CALL_KERNEL(spParallelDeviceFillIntKernel, 16, 256, d, v, s);

    return SP_SUCCESS;
};

__global__
void spParallelDeviceFillRealKernel(Real *d, Real v, size_type max)
{
    for (size_type s = threadIdx.x + blockIdx.x * blockDim.x; s < max; s += gridDim.x * blockDim.x) { d[s] = v; }
};

int spParallelDeviceFillReal(Real *d, Real v, size_type s)
{
    SP_DEVICE_CALL_KERNEL(spParallelDeviceFillRealKernel, 16, 256, d, v, s);
    return SP_SUCCESS;
};


__global__
void spParallelAssignKernel(size_type max, size_type const *offset, Real *d, Real const *v)
{

    size_type num_of_thread = blockDim.x * gridDim.x * blockDim.x * gridDim.x * blockDim.x * gridDim.x;

    for (size_type s = (threadIdx.x + blockIdx.x * blockDim.x) +
        (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x +
        (threadIdx.x + blockIdx.x * blockDim.x) * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
         s < max; s += num_of_thread) { d[offset[s]] = v[s]; }
};

int spParallelAssign(size_type num_of_point, size_type *offset, Real *d, Real const *v)
{
    SP_DEVICE_CALL_KERNEL(spParallelAssignKernel, 16, 256, num_of_point, offset, d, v);
    return SP_SUCCESS;
};

int spMemoryDeviceToHost(void **p, void *src, size_type size_in_byte)
{
    return SP_CALL(spParallelHostAlloc(p, size_in_byte)) ||
        SP_CALL(spParallelMemcpy(*p, src, size_in_byte));

}
int spMemoryHostFree(void **p)
{
    return SP_CALL(spParallelHostFree(p));

}