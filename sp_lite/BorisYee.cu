#include "hip/hip_runtime.h"
//
// Created by salmon on 16-6-14.
//

#include "Boris.h"
#include <stdio.h>
#include <assert.h>

#include "sp_def.h"
#include "spField.h"
#include "spMesh.h"
#include "spParticle.h"
#include "spPage.h"

//#include "spMesh.cu"
//#include "spField.cu"
//#include "spParticle.cu"

/******************************************************************************************/

__global__ void spInitializeParticle_BorisYee_Kernel(spMesh *ctx, spPage** buckets, spPage * pool,
		size_type entity_size_in_byte)
{

	index_type g_x = (blockIdx.x);
	index_type g_y = (blockIdx.y);
	index_type g_z = (blockIdx.z);

	size_type g_dim_x = gridDim.x;
	size_type g_dim_y = gridDim.y;
	size_type g_dim_z = gridDim.z;

	index_type t_x = (threadIdx.x);
	index_type t_y = (threadIdx.y);
	index_type t_z = (threadIdx.z);
	size_type t_dim_x = blockDim.x;
	size_type t_dim_y = blockDim.y;
	size_type t_dim_z = blockDim.z;

	size_type g_num = g_x + (g_y + g_z * g_dim_y) * g_dim_x;
	size_type t_num = t_x + (t_y + t_z * t_dim_y) * t_dim_x;

	buckets[g_num] = &pool[g_num];
	if (t_num == 0)
	{
		buckets[g_num]->next = 0x0;
		buckets[g_num]->flag = ~0x0;
	}

	boris_point_s*p = (boris_point_s*) (buckets[g_num]->data + (t_num * entity_size_in_byte));

	p->tag = 0;
	p->r[0] = 0.5;
	p->r[1] = 0.5;
	p->r[2] = 0.5;

	p->v[0] = 0.5;
	p->v[1] = 0.5;
	p->v[2] = 0.5;

	p->f = 1.0;
	p->w = 1.0;

}

void spInitializeParticle_BorisYee(spMesh *ctx, sp_particle_type *sp, size_type NUM_OF_PIC)
{

//	hipStream_t s_shared[ctx->number_of_shared_blocks];
//
//	for (int i = 0, ie = ctx->number_of_shared_blocks; i < ie; ++i)
//	{
//		hipStreamCreate(&s_shared[i]);
//		spInitializeParticle_BorisYee_Kernel<<<ctx->shared_blocks[i], ctx->threadsPerBlock, 0, s_shared[i]>>>(
//				(spMesh *) spObject_device_((spObject*) ctx), (sp_particle_type *) spObject_device_((spObject*) pg),
//				NUM_OF_PIC);
//	}

	hipStream_t s_local;
	hipStreamCreate(&s_local);
	spInitializeParticle_BorisYee_Kernel<<<ctx->private_block, ctx->threadsPerBlock, 0, s_local>>>(
			(spMesh *) spObject_device_((spObject*) ctx), sp->buckets, sp->m_pages, sp->entity_size_in_byte);

//	for (int i = 0, ie = ctx->number_of_shared_blocks; i < ie; ++i)
//	{
//		hipStreamSynchronize(s_shared[i]); //wait for boundary
//
//	}
//
//	spSyncParticle(ctx, pg);

	hipDeviceSynchronize(); //wait for iteration to finish
}

/******************************************************************************************/

//__device__ double atomicAddD(double* address, double val)
//{
//	unsigned long long int* address_as_ull = (unsigned long long int*) address;
//	unsigned long long int old = *address_as_ull, assumed;
//	do
//	{
//		assumed = old;
//		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
//		// Note: uses integer comparison to avoid hang in case of NaN (since NaN !=		NaN	)
//	} while (assumed != old);
//	return __longlong_as_double(old);
//}
#define ll 0
#define rr 1.0
#define RADIUS 2
#define CACHE_EXTENT_X RADIUS*2
#define CACHE_EXTENT_Y RADIUS*2
#define CACHE_EXTENT_Z RADIUS*2
#define CACHE_SIZE (CACHE_EXTENT_X*CACHE_EXTENT_Y*CACHE_EXTENT_Z)
#define IX  1
#define IY  CACHE_EXTENT_X
#define IZ  CACHE_EXTENT_X*CACHE_EXTENT_Y
__device__
void cache_gather(Real *v, Real const *f, Real rx, Real ry, Real rz)
{

	*v = f[ IX + IY + IZ /*  */] * (rx - ll) * (ry - ll) * (rz - ll)
			+ f[ IX + IY /*     */] * (rx - ll) * (ry - ll) * (rr - rz)
			+ f[ IX + IZ /*     */] * (rx - ll) * (rr - ry) * (rz - ll)
			+ f[ IX /*          */] * (rx - ll) * (rr - ry) * (rr - rz)
			+ f[ IY + IZ /*     */] * (rr - rx) * (ry - ll) * (rz - ll)
			+ f[ IY /*          */] * (rr - rx) * (ry - ll) * (rr - rz)
			+ f[ IZ /*          */] * (rr - rx) * (rr - ry) * (rz - ll)
			+ f[0 /*               */] * (rr - rx) * (rr - ry) * (rr - rz);
}
__device__
void cache_scatter(Real *f, Real v, Real rx, Real ry, Real rz)
{
	atomicAdd(&(f[ IX + IY + IZ /*  */]), v * (rx - ll) * (ry - ll) * (rz - ll));
	atomicAdd(&(f[ IX + IY /*       */]), v * (rx - ll) * (ry - ll) * (rr - rz));
	atomicAdd(&(f[ IX + IZ /*       */]), v * (rx - ll) * (rr - ry) * (rz - ll));
	atomicAdd(&(f[ IX /*            */]), v * (rx - ll) * (rr - ry) * (rr - rz));
	atomicAdd(&(f[ IY + IZ /*       */]), v * (rr - rx) * (ry - ll) * (rz - ll));
	atomicAdd(&(f[ IY /*            */]), v * (rr - rx) * (ry - ll) * (rr - rz));
	atomicAdd(&(f[ IZ /*            */]), v * (rr - rx) * (rr - ry) * (rz - ll));
	atomicAdd(&(f[0 /*              */]), v * (rr - rx) * (rr - ry) * (rr - rz));
}

//#undef ll
//#undef rr
//#undef IX
//#undef IY
//#undef IZ
#define _R 1.0
MC_CONSTANT Real id_to_shift_[][3] =
{ //
		{ 0, 0, 0 },           // 000
				{ _R, 0, 0 },           // 001
				{ 0, _R, 0 },           // 010
				{ 0, 0, _R },          // 011
				{ _R, _R, 0 },           // 100
				{ _R, 0, _R },          // 101
				{ 0, _R, _R },          // 110
				{ 0, _R, _R },          // 111
		};
MC_CONSTANT int sub_index_to_id_[4][3] =
{ //
		{ 0, 0, 0 }, /*VERTEX*/
		{ 1, 2, 4 }, /*EDGE*/
		{ 6, 5, 3 }, /*FACE*/
		{ 7, 7, 7 } /*VOLUME*/

		};
MC_CONSTANT int cache_cell_offset_tag[CACHE_SIZE] =
{ };
MC_CONSTANT size_type cache_cell_offset[CACHE_SIZE] =
{ };
#undef _R
//
//__device__
//inline void spBorisPushOne(struct boris_point_s const *p0, struct boris_point_s *p1, Real dt, Real q, Real m,
//		Real const * tE, Real const * tB, Real * tJ, const float3 inv_dx)
//{
//
//	Real E[3], B[3];
//
//	cache_gather(&E[0], tE + CACHE_SIZE * 0, p0->r); //, id_to_shift_[sub_index_to_id_[1/*EDGE*/][0]]);
//	cache_gather(&E[1], tE + CACHE_SIZE * 1, p0->r); //, id_to_shift_[sub_index_to_id_[1/*EDGE*/][1]]);
//	cache_gather(&E[2], tE + CACHE_SIZE * 2, p0->r); //, id_to_shift_[sub_index_to_id_[1/*EDGE*/][2]]);
//
//	cache_gather(&B[0], tB + CACHE_SIZE * 0, p0->r); //, id_to_shift_[sub_index_to_id_[2/*FACE*/][0]]);
//	cache_gather(&B[1], tB + CACHE_SIZE * 1, p0->r); //, id_to_shift_[sub_index_to_id_[2/*FACE*/][1]]);
//	cache_gather(&B[2], tB + CACHE_SIZE * 2, p0->r); //, id_to_shift_[sub_index_to_id_[2/*FACE*/][2]]);
//
//	p1->r[0] = p0->r[0] + p0->v[0] * dt * 0.5 * inv_dx.x;
//	p1->r[1] = p0->r[1] + p0->v[1] * dt * 0.5 * inv_dx.y;
//	p1->r[2] = p0->r[2] + p0->v[2] * dt * 0.5 * inv_dx.z;
//
//	Real v_[3], t[3];
//
//	t[0] = B[0] * (q / m * dt * 0.5);
//	t[1] = B[1] * (q / m * dt * 0.5);
//	t[2] = B[2] * (q / m * dt * 0.5);
//
//	p1->v[0] = p0->v[0] + E[0] * (q / m * dt * 0.5);
//	p1->v[1] = p0->v[1] + E[1] * (q / m * dt * 0.5);
//	p1->v[2] = p0->v[2] + E[2] * (q / m * dt * 0.5);
//
//	v_[0] = p1->v[0] + (p1->v[1] * t[2] - p1->v[2] * t[1]);
//	v_[1] = p1->v[1] + (p1->v[2] * t[0] - p1->v[0] * t[2]);
//	v_[2] = p1->v[2] + (p1->v[0] * t[1] - p1->v[1] * t[0]);
//
//	Real tt = t[0] * t[0] + t[1] * t[1] + t[2] * t[2] + 1.0;
//
//	p1->v[0] += (v_[1] * t[2] - v_[2] * t[1]) * 2.0 / tt;
//	p1->v[1] += (v_[2] * t[0] - v_[0] * t[2]) * 2.0 / tt;
//	p1->v[2] += (v_[0] * t[1] - v_[1] * t[0]) * 2.0 / tt;
//
//	p1->v[0] += E[0] * (q / m * dt * 0.5);
//	p1->v[1] += E[1] * (q / m * dt * 0.5);
//	p1->v[2] += E[2] * (q / m * dt * 0.5);
//
//	p1->r[0] += p1->v[0] * dt * 0.5 * inv_dx.x;
//	p1->r[1] += p1->v[1] * dt * 0.5 * inv_dx.y;
//	p1->r[2] += p1->v[2] * dt * 0.5 * inv_dx.z;
//
//	cache_scatter(tJ + CACHE_SIZE * 0, p1->f * p1->w * q, p1->r); //, id_to_shift_[sub_index_to_id_[0/*VERTEX*/][0]]);
//	cache_scatter(tJ + CACHE_SIZE * 1, p1->f * p1->w * p1->v[0] * q, p1->r); //, id_to_shift_[sub_index_to_id_[1/*EDGE*/][0]]);
//	cache_scatter(tJ + CACHE_SIZE * 2, p1->f * p1->w * p1->v[1] * q, p1->r); //, id_to_shift_[sub_index_to_id_[1/*EDGE*/][1]]);
//	cache_scatter(tJ + CACHE_SIZE * 3, p1->f * p1->w * p1->v[2] * q, p1->r); //, id_to_shift_[sub_index_to_id_[1/*EDGE*/][2]]);
//
//}

/******************************************************************************************/
#define NUMBER_OF_THREAD 128

//#define DISABLE_SOA

__global__ void spUpdateParticle_BorisYee_Kernel(spMesh *m, Real dt, Real charge, Real mass, //
		int lower_x, int lower_y, int lower_z, int upper_x, int upper_y, int upper_z, //
		size_type entity_size_in_byte, spPage** buckets, const Real *fE, const Real *fB, Real *fRho, Real *fJ)
{

	int dim_x = m->dims.x;
	int dim_y = m->dims.y;
	int dim_z = m->dims.z;
	int total_g_num = dim_x * dim_y * dim_z;

	const int t_num = threadIdx.x + (threadIdx.y + threadIdx.z * blockDim.y) * blockDim.x;

	const int num_t = blockDim.x * blockDim.y * blockDim.z;

	const Real inv_dx = m->inv_dx.x;
	const Real inv_dy = m->inv_dx.y;
	const Real inv_dz = m->inv_dx.z;

	__shared__ Real tE[24], tB[3 * 8], tJ[4 * 8];

	for (int g_x = (lower_x + (blockIdx.x * (upper_x - lower_x)) / gridDim.x), //
			x_upper = (lower_x + ((blockIdx.x + 1) * (upper_x - lower_x)) / gridDim.x); g_x < x_upper; ++g_x)
		for (int g_y = (lower_y + (blockIdx.y * (upper_y - lower_y)) / gridDim.y), //
				y_upper = (lower_y + ((blockIdx.y + 1) * (upper_y - lower_y)) / gridDim.y); g_y < y_upper; ++g_y)
			for (int g_z = (lower_z + (blockIdx.z * (upper_z - lower_z)) / gridDim.z), //
					z_upper = (lower_z + ((blockIdx.z + 1) * (upper_z - lower_z)) / gridDim.z); g_z < z_upper; ++g_z)
			{
				int g_num = g_x + (g_y + g_z * dim_y) * dim_x;

				assert(g_num < total_g_num);
//
				int g_f_x = (g_x + threadIdx.x + dim_x - RADIUS) % dim_x;
				int g_f_y = (g_y + threadIdx.y + dim_y - RADIUS) % dim_y;
				int g_f_z = (g_z + threadIdx.z + dim_z - RADIUS) % dim_z;
				int g_f_num = g_f_x + (g_f_y + g_f_z * dim_y) * dim_x;

//				if (threadIdx.x < CACHE_EXTENT_X && threadIdx.y < CACHE_EXTENT_Y && threadIdx.z < CACHE_EXTENT_Y)
				{
//					tE[0][threadIdx.x % blockDim.x][threadIdx.y % blockDim.y][threadIdx.z % blockDim.z] = fE[g_f_num * 3
//							+ 0];
//					tE[1][threadIdx.x % blockDim.x][threadIdx.y % blockDim.y][threadIdx.z % blockDim.z] = fE[g_f_num * 3
//							+ 0];
//					tE[2][threadIdx.x % blockDim.x][threadIdx.y % blockDim.y][threadIdx.z % blockDim.z] = fE[g_f_num * 3
//							+ 0];

//					tE[0 * CACHE_SIZE + t_num] = fE[g_f_num * 3 + 0];
//					tE[1 * CACHE_SIZE + t_num] = fE[g_f_num * 3 + 1];
//					tE[2 * CACHE_SIZE + t_num] = fE[g_f_num * 3 + 2];
//
//					tB[0 * CACHE_SIZE + t_num] = fB[g_f_num * 3 + 0];
//					tB[1 * CACHE_SIZE + t_num] = fB[g_f_num * 3 + 1];
//					tB[2 * CACHE_SIZE + t_num] = fB[g_f_num * 3 + 2];
//
//					tJ[0 * CACHE_SIZE + t_num] = 0;
//					tJ[1 * CACHE_SIZE + t_num] = 0;
//					tJ[2 * CACHE_SIZE + t_num] = 0;
//					tJ[3 * CACHE_SIZE + t_num] = 0;

				}

				__syncthreads();

				spPage *src = buckets[g_f_num];
				Real beta = (charge / mass * dt * 0.5);

				while (src != 0x0)
				{

#ifdef ENABLE_SOA

					struct boris_page_s * pd = (struct boris_page_s *) src->data;

#endif

					for (int s = t_num; s < SP_NUMBER_OF_ENTITIES_IN_PAGE; s += num_t)
					{
#ifdef ENABLE_SOA

						Real rx = pd->r[0][s], ry = pd->r[1][s], rz = pd->r[2][s];

						Real vx = pd->v[0][s], vy = pd->v[1][s], vz = pd->v[2][s];

						Real f = pd->f[s], w = pd->w[s];

#else

						struct boris_point_s *p = (struct boris_point_s *) (src->data + s * entity_size_in_byte);

						Real rx = p->r[0], ry = p->r[1], rz = p->r[2];

						Real vx = p->v[0], vy = p->v[1], vz = p->v[2];

						Real f = p->f, w = p->w;
#endif
						Real v_x, v_y, v_z;

						Real tx, ty, tz;

						Real Bx, By, Bz, Ex, Ey, Ez;

						cache_gather(&Ex, tE + 8 * 0, rx, ry, rz); //, id_to_shift_[sub_index_to_id_[1/*EDGE*/][0]]);
						cache_gather(&Ey, tE + 8 * 1, rx, ry, rz); //, id_to_shift_[sub_index_to_id_[1/*EDGE*/][1]]);
						cache_gather(&Ez, tE + 8 * 2, rx, ry, rz); //, id_to_shift_[sub_index_to_id_[1/*EDGE*/][2]]);

						cache_gather(&Bx, tB + 8 * 0, rx, ry, rz); //, id_to_shift_[sub_index_to_id_[2/*FACE*/][0]]);
						cache_gather(&By, tB + 8 * 1, rx, ry, rz); //, id_to_shift_[sub_index_to_id_[2/*FACE*/][1]]);
						cache_gather(&Bz, tB + 8 * 2, rx, ry, rz); //, id_to_shift_[sub_index_to_id_[2/*FACE*/][2]]);

						rx += vx * dt * 0.5 * inv_dx;
						ry += vy * dt * 0.5 * inv_dy;
						rz += vz * dt * 0.5 * inv_dz;

						tx = Bx * beta;
						ty = By * beta;
						tz = Bz * beta;

						vx += Ex * beta;
						vy += Ey * beta;
						vz += Ez * beta;

						v_x = vx + (vy * tz - vz * ty);
						v_y = vy + (vz * tx - vx * tz);
						v_z = vz + (vx * ty - vy * tx);

						Real tt = 2.0 / (tx * tx + ty * ty + tz * tz + 1.0);

						vx += (v_y * tz - v_z * ty) * tt;
						vy += (v_z * tx - v_x * tz) * tt;
						vz += (v_x * ty - v_y * tx) * tt;

						vx += Ex * beta;
						vy += Ey * beta;
						vz += Ez * beta;

						rx += vx * dt * 0.5 * inv_dx;
						ry += vy * dt * 0.5 * inv_dy;
						rz += vz * dt * 0.5 * inv_dz;

//						cache_scatter(tJ + 8 * 0, f * w * charge, rx, ry, rz); //, id_to_shift_[sub_index_to_id_[0/*VERTEX*/]x[o+t_num]]);
//						cache_scatter(tJ + 8 * 1, f * w * vx * charge, rx, ry, rz); //, id_to_shift_[sub_index_to_id_[1/*EDGE*/]x[o+t_num]]);
//						cache_scatter(tJ + 8 * 2, f * w * vy * charge, rx, ry, rz); //, id_to_shift_[sub_index_to_id_[1/*EDGE*/][1]]);
//						cache_scatter(tJ + 8 * 3, f * w * vz * charge, rx, ry, rz); //, id_to_shift_[sub_index_to_id_[1/*EDGE*/][2]]);
#ifdef ENABLE_SOA
						pd->r[0][s] = rx;
						pd->r[1][s] = ry;
						pd->r[2][s] = rz;

						pd->v[0][s] = vx;
						pd->v[1][s] = vy;
						pd->v[2][s] = vz;
#else
						p->r[0] = rx;
						p->r[1] = ry;
						p->r[2] = rz;

						p->v[0] = vx;
						p->v[1] = vy;
						p->v[2] = vz;
#endif

					}
					__syncthreads();

					src = src->next;
				}        //	for (int n = 0; n < CACHE_SIZE; ++n)

			}
//	atomicAdd(&(fRho[g_f_num]), ttJ[0][t_num]);
//	atomicAdd(&(fJ[g_f_num * 3 + 0]), ttJ[1][t_num]);
//	atomicAdd(&(fJ[g_f_num * 3 + 1]), ttJ[2][t_num]);
//	atomicAdd(&(fJ[g_f_num * 3 + 2]), ttJ[3][t_num]);
}

//		if (t_num == 0 && g_f_num == 0)
//		{
//			printf("page flag = %x,  tag=%x, r=[%d,%d,%d], v=[%d,%d,%d], f= %d, w=%d, \n",
//					src->flag, //
//					((struct boris_point_s const *) p0)->tag, ((struct boris_point_s const *) p0)->r[0],
//					((struct boris_point_s const *) p0)->r[1], ((struct boris_point_s const *) p0)->r[2],
//					((struct boris_point_s const *) p0)->v[0], ((struct boris_point_s const *) p0)->v[1],
//					((struct boris_point_s const *) p0)->v[2], ((struct boris_point_s const *) p0)->f,
//					((struct boris_point_s const *) p0)->w);
//		}
void spUpdateParticle_BorisYee(spMesh *ctx, Real dt, sp_particle_type *pg, const sp_field_type *fE,
		const sp_field_type *fB, sp_field_type *fRho, sp_field_type *fJ)
{
//	hipStream_t s1;
//	hipStreamCreate(&s1);
//	hipStream_t s2;
//	hipStreamCreate(&s2);
//
//	spUpdateParticle_BorisYee_Kernel<<<ctx->numBlocks, ctx->threadsPerBlock, 0, s1>>>(
//			(spMesh *) spObject_device_((spObject*) ctx),
//			dt, //
//			(sp_particle_type *) spObject_device_((spObject*) pg),
//			(const sp_field_type *) spObject_device_((spObject*) fE),
//			(const sp_field_type *) spObject_device_((spObject*) fB),
//			(sp_field_type *) spObject_device_((spObject*) fRho), (sp_field_type *) spObject_device_((spObject*) fJ));
	dim3 grid_dim;
	grid_dim.x = 8;
	grid_dim.y = 8;
	grid_dim.z = 4;

	spUpdateParticle_BorisYee_Kernel<<<grid_dim, NUMBER_OF_THREAD>>>( //
			(spMesh*) (ctx->device_self), //
			dt, pg->charge, pg->mass, //
			ctx->x_lower.x, ctx->x_lower.y, ctx->x_lower.z, //
			ctx->x_upper.x, ctx->x_upper.y, ctx->x_upper.z, //
			pg->entity_size_in_byte, //
			pg->buckets, //
			((Real*) fE->device_data), //
			((Real*) fB->device_data), //
			((Real*) fRho->device_data), //
			((Real*) fJ->device_data) //
			);

//	spSyncParticle(ctx, pg);
//	spSyncField(ctx, fJ);
//	spSyncField(ctx, fRho);
	hipDeviceSynchronize(); //wait for iteration to finish

}
/***************************************************************************************************************/
//__global__ void spUpdateField_Yee_kernel(spMesh *ctx, Real dt, const sp_field_type *fRho, const sp_field_type *fJ,
//		sp_field_type *fE, sp_field_type *fB)
__global__ void spUpdateField_Yee_kernel(spMesh *ctx, Real dt, const Real *fRho, const Real *fJ, Real *fE, Real *fB)
{
	index_type ix = (blockIdx.x * blockDim.x + threadIdx.x);
	index_type iy = (blockIdx.y * blockDim.y + threadIdx.y);
	index_type iz = (blockIdx.z * blockDim.z + threadIdx.z);
	size_type dim_x = gridDim.x * blockDim.x;
	size_type dim_y = gridDim.y * blockDim.y;
	size_type dim_z = gridDim.z * blockDim.z;

	int n = ix + (iy + iz * dim_y) * dim_z;

	(fE)[n * 3 + 0] = ix;
	(fE)[n * 3 + 1] = iy;
	(fE)[n * 3 + 2] = iz;

}
void spUpdateField_Yee(spMesh *ctx, Real dt, const sp_field_type *fRho, const sp_field_type *fJ, sp_field_type *fE,
		sp_field_type *fB)
{

//	hipStream_t s_shared[ctx->number_of_shared_blocks];
//
//	for (int i = 0, ie = ctx->number_of_shared_blocks; i < ie; ++i)
//	{
//		hipStreamCreate(&s_shared[i]);
//
//		spUpdateField_Yee_kernel<<<ctx->shared_blocks[i], ctx->threadsPerBlock, 0, s_shared[i]>>>(
//				(spMesh *) spObject_device_((spObject*) ctx),
//				dt, //
//				(const sp_field_type *) spObject_device_((spObject*) fRho),
//				(const sp_field_type *) spObject_device_((spObject*) fJ),
//				(sp_field_type *) spObject_device_((spObject*) fE), (sp_field_type *) spObject_device_((spObject*) fB));
//	}
//	hipStream_t s_local;
//	hipStreamCreate(&s_local);
//
//	spUpdateField_Yee_kernel<<<ctx->private_block, 1>>>((spMesh *) spObject_device_((spObject*) ctx),
//			dt, //
//			(const sp_field_type *) spObject_device_((spObject*) fRho),
//			(const sp_field_type *) spObject_device_((spObject*) fJ),
//			(sp_field_type *) spObject_device_((spObject*) fE), (sp_field_type *) spObject_device_((spObject*) fB));

	dim3 grid_dim;
	grid_dim.x = 4;
	grid_dim.y = 4;
	grid_dim.z = 4;
	dim3 t_per_block;
	t_per_block.x = 8;
	t_per_block.y = 4;
	t_per_block.z = 4;
//	grid_dim.x = ctx->private_block.x / ctx->threadsPerBlock.x;
//	grid_dim.y = ctx->private_block.y / ctx->threadsPerBlock.y;
//	grid_dim.z = ctx->private_block.z / ctx->threadsPerBlock.z;
	spUpdateField_Yee_kernel<<<grid_dim, t_per_block>>>( //
			(spMesh*) (ctx->device_self), dt,        //
			((Real*) fRho->device_data), //
			((Real*) fJ->device_data), //
			((Real*) fE->device_data), ((Real*) fB->device_data));
//	for (int i = 0, ie = ctx->number_of_shared_blocks; i < ie; ++i)
//	{
//		hipStreamSynchronize(s_shared[i]); //wait for boundary
//	}
//
//	spSyncField(ctx, fE);
//	spSyncField(ctx, fB);

	hipDeviceSynchronize();        //wait for iteration to finish

}

