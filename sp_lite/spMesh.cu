#include "hip/hip_runtime.h"
/*
 * spMesh.c
 *
 *  Created on: 2016年6月15日
 *      Author: salmon
 */
#include <assert.h>
#include "sp_def.h"
#include "spMesh.h"

void spMeshCreate(spMesh **ctx)
{
	*ctx = (spMesh *) malloc(sizeof(spMesh));

}
void spMeshDestroy(spMesh **ctx)
{
	free(*ctx);
}
void spMeshDeploy(spMesh *self)
{
	self->ndims = 3;
	self->offset.x = 0;
	self->offset.y = 0;
	self->offset.z = 0;
	self->offset.w = 0;
	self->count.x = self->dims.x;
	self->count.y = self->dims.y;
	self->count.z = self->dims.z;
	self->x_lower = 0;
	self->x_upper.x = self->dims.x;
	self->x_upper.y = self->dims.y;
	self->x_upper.z = self->dims.z;

//	self->dims.w = 3;
//	self->offset.w = 0;
//	self->count.w = 3;

	self->threadsPerBlock.x = 4;
	self->threadsPerBlock.y = 4;
	self->threadsPerBlock.z = 4;

	self->number_of_shared_blocks = 0;
	self->private_block.x = self->dims.x;
	self->private_block.y = self->dims.y;
	self->private_block.z = self->dims.z;

	/**          -1
	 *
	 *    -1     0    1
	 *
	 *           1
	 */
	/**
	 *\verbatim
	 *                ^y
	 *               /
	 *        z     /
	 *        ^    /
	 *    PIXEL0 110-------------111 VOXEL
	 *        |  /|              /|
	 *        | / |             / |
	 *        |/  |    PIXEL1  /  |
	 * EDGE2 100--|----------101  |
	 *        | m |           |   |
	 *        |  010----------|--011 PIXEL2
	 *        |  / EDGE1      |  /
	 *        | /             | /
	 *        |/              |/
	 *       000-------------001---> x
	 *                       EDGE0
	 *
	 *\endverbatim
	 */
	int3 neighbour_offset[27];
	int neighbour_flag[27];
	int count = 0;
	for (int i = -1; i <= 1; ++i)
		for (int j = -1; j <= 1; ++j)
			for (int k = -1; k <= 1; ++k)
			{
				neighbour_offset[count].x = i;
				neighbour_offset[count].y = j;
				neighbour_offset[count].z = k;
				neighbour_flag[count] = (i + 1) | ((j + 1) << 2) | ((k + 1) << 4);
				++count;
			}
	assert(count == 27);
	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(SP_NEIGHBOUR_OFFSET), neighbour_offset, sizeof(neighbour_offset)));
	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(SP_NEIGHBOUR_OFFSET_flag), neighbour_flag, sizeof(neighbour_flag)));

}

size_type spMeshGetNumberOfEntity(spMesh const *self, int iform)
{
	return self->dims.x * self->dims.y * self->dims.z * ((iform == 0 || iform == 3) ? 1 : 3);
}

