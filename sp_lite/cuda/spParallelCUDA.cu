#include "hip/hip_runtime.h"
//
// Created by salmon on 16-7-25.
//

extern "C" {
#include "spParallelCUDA.h"
}
dim3 sizeType2Dim3(size_type const *v)
{
    dim3 res;
    res.x = (int) v[0];
    res.y = (int) v[1];
    res.z = (int) v[2];
    return res;
}
Real3 real2Real3(Real const *v)
{
    Real3 res;
    res.x = (Real) v[0];
    res.y = (Real) v[1];
    res.z = (Real) v[2];
    return res;
}

int spParallelDeviceInitialize(int argc, char **argv)
{
    int num_of_device = 0;
    SP_CUDA_CALL(hipGetDeviceCount(&num_of_device));
    SP_CUDA_CALL(hipSetDevice(spMPIRank() % num_of_device));
    SP_CUDA_CALL(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
    SP_CUDA_CALL(hipGetLastError());
}

int spParallelDeviceFinalize()
{
    SP_CUDA_CALL(hipDeviceReset());
}

int spParallelDeviceAlloc(void **p, size_type s)
{
    SP_CUDA_CALL(hipMalloc(p, s));
    return SP_SUCCESS;
}

int spParallelDeviceFree(void **_P_)
{
    if (*_P_ != NULL)
    {
        SP_CUDA_CALL(hipFree(*_P_));
        *_P_ = NULL;
    }
    return SP_SUCCESS;
};

int spParallelMemcpy(void *dest, void const *src, size_type s)
{
    SP_CUDA_CALL(hipMemcpy(dest, src, s, hipMemcpyDefault));
    return SP_SUCCESS;
}

int spParallelMemcpyToSymbol(void **dest, void const **src, size_type s)
{
    SP_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dest), src, s, hipMemcpyDefault));
    return SP_SUCCESS;
}

int spParallelMemset(void *dest, int v, size_type s)
{
    SP_CUDA_CALL(hipMemset(dest, v, s));
    return SP_SUCCESS;
}

int spParallelDeviceSync()
{
    SP_CALL(spParallelGlobalBarrier());
    SP_CUDA_CALL(hipDeviceSynchronize());
    return SP_SUCCESS;
}

int spParallelHostAlloc(void **p, size_type s)
{
    SP_CUDA_CALL(hipHostAlloc(p, s, hipHostMallocDefault));
    return SP_SUCCESS;
};

int spParallelHostFree(void **p)
{
    if (*p != NULL)
    {
        hipHostFree(*p);
        *p = NULL;
    }
    return SP_SUCCESS;
}


__global__
void spParallelDeviceFillIntKernel(int *d, int v, size_type max)
{
    for (size_t s = threadIdx.x + blockIdx.x * blockDim.x; s < max; s += gridDim.x * blockDim.x) { d[s] = v; }
};
int spParallelDeviceFillInt(int *d, int v, size_type s)
{
    CALL_KERNEL(spParallelDeviceFillIntKernel, 16, 256, d, v, s);

    return SP_SUCCESS;
};

__global__
void spParallelDeviceFillRealKernel(Real *d, Real v, size_type max)
{
    for (size_type s = threadIdx.x + blockIdx.x * blockDim.x; s < max; s += gridDim.x * blockDim.x) { d[s] = v; }
};
int spParallelDeviceFillReal(Real *d, Real v, size_type s)
{
    CALL_KERNEL(spParallelDeviceFillRealKernel, 16, 256, d, v, s);
    return SP_SUCCESS;
};


__global__
void spParallelAssignKernel(size_type max, size_type const *offset, Real *d, Real const *v)
{

    size_type num_of_thread = blockDim.x * gridDim.x * blockDim.x * gridDim.x * blockDim.x * gridDim.x;

    for (size_type s = (threadIdx.x + blockIdx.x * blockDim.x) +
        (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x +
        (threadIdx.x + blockIdx.x * blockDim.x) * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
         s < max; s += num_of_thread) { d[offset[s]] = v[s]; }
};

int spParallelAssign(size_type num_of_point, size_type *offset, Real *d, Real const *v)
{
    CALL_KERNEL(spParallelAssignKernel, 16, 256, num_of_point, offset, d, v);
    return SP_SUCCESS;
};


int spRandomUniformN(Real **data, int num_of_dims, size_type num_of_sample, Real const *lower, Real const *upper)
{
    return SP_SUCCESS;
};

/**
 *  \f[
 *      f\left(v\right)\equiv\frac{1}{\sqrt{\left(2\pi\sigma\right)^{3}}}\exp\left(-\frac{\left(v-u\right)^{2}}{\sigma^{2}}\right)
 *  \f]
 * @param data
 * @param num_of_sample
 * @param u0
 * @param sigma
 * @return
 */
int spRandomNormal3(Real **data, size_type num_of_sample, Real const *u0, Real sigma)
{
    return SP_SUCCESS;

}