/*
 * spField.cu
 *
 *  Created on: 2016年6月15日
 *      Author: salmon
 */
#include "sp_def.h"
#include "spMesh.h"
#include "spField.h"
#include "spObject.h"
#include "spSimPlaWrap.h"

MC_HOST void spCreateField(const spMesh *mesh, sp_field_type **f, int iform)
{
	spCreateObject((spObject **) f, sizeof(sp_field_type));
//	*f = (sp_field_type *) malloc(sizeof(sp_field_type));
	(*f)->iform = iform;
	(*f)->number_of_entities = spMeshGetNumberOfEntity(mesh, iform);

	CUDA_CHECK_RETURN(hipMalloc((void ** ) &((*f)->data), (*f)->number_of_entities * sizeof(Real)));

}

MC_HOST void spDestroyField(sp_field_type **f)
{
	if (f != 0x0 && *f != 0x0)
	{
		spFree((void **) &((**f).data));
		spDestroyObject((spObject **) f);
	}
	*f = 0x0;
}
MC_HOST int spWriteField(spMesh const *mesh, sp_field_type const *f, char const url[], int flag)
{
	size_type num_of_entity = spMeshGetNumberOfEntity(mesh, f->iform);
	CUDA_CHECK(num_of_entity);
	Real * tmp = (Real*) malloc(num_of_entity * sizeof(Real));

	CUDA_CHECK_RETURN(hipMemcpy(tmp, (void* )(f->data), num_of_entity * sizeof(Real), hipMemcpyDeviceToHost));
	for (size_type s = 0; s < num_of_entity; ++s)
	{
		printf(" %f ", (tmp[s]));

	}
	int ndims = (f->iform == 1 || f->iform == 2) ? 4 : 3;

	hdf5_write_field(url, (void*) tmp, ndims, mesh->dims, mesh->offset, mesh->count, flag);
	free(tmp);
	return 0;
}
MC_HOST int spSyncField(spMesh const *mesh, sp_field_type *f)
{

}
