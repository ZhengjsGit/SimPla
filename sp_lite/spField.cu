/*
 * spField.cu
 *
 *  Created on: 2016年6月15日
 *      Author: salmon
 */
#include "sp_def.h"
#include "spMesh.h"
#include "spField.h"
#include "spObject.h"
#include "spSimPlaWrap.h"

MC_HOST void spCreateField(const spMesh *mesh, sp_field_type **f, int iform)
{
	spCreateObject((spObject **) f, sizeof(sp_field_type));
//	*f = (sp_field_type *) malloc(sizeof(sp_field_type));
	(*f)->iform = iform;
	(*f)->host_data = 0x0;
	(*f)->device_data = 0x0;

	size_type num_of_entities = spMeshGetNumberOfEntity(mesh, iform);

	CUDA_CHECK_RETURN(hipMalloc((void ** ) &((*f)->device_data), num_of_entities * sizeof(Real)));

	(*f)->host_data = (Real*) malloc(num_of_entities * sizeof(Real));

}

MC_HOST void spDestroyField(sp_field_type **f)
{
	if (f != 0x0 && *f != 0x0)
	{
		if ((**f).device_data != 0x0)
		{
			CUDA_CHECK_RETURN(hipFree((void** )((**f).device_data)))
		};

		if ((**f).host_data != 0x0)
		{
			free((void**) ((**f).host_data));
		}
		*f = 0x0;
	}
}
MC_HOST void spClearField(spMesh const *mesh, sp_field_type *f)
{
	size_type num_of_entities = spMeshGetNumberOfEntity(mesh, f->iform);

	if (f->device_data != 0x0)
	{
		CUDA_CHECK_RETURN(hipMemset(f->device_data, 0, num_of_entities * sizeof(Real)));
	}
}

MC_HOST int spWriteField(spMesh const *mesh, sp_field_type *f, char const url[], int flag)
{
	size_type num_of_entities = spMeshGetNumberOfEntity(mesh, f->iform);

//	CUDA_CHECK_RETURN(
	hipMemcpy((void*) (f->host_data), (void*) (f->device_data), num_of_entities * sizeof(Real), hipMemcpyDefault)
//	)
			;
	int ndims = (f->iform == 1 || f->iform == 2) ? 4 : 3;
	hdf5_write_field(url, (void*) f->host_data, ndims, mesh->dims, mesh->offset, mesh->count, flag);
	return 0;
}
MC_HOST int spSyncField(spMesh const *mesh, sp_field_type *f)
{

}
