/*
 * BorisYeeMain.c
 *
 *  Created on: 2016年6月15日
 *      Author: salmon
 */

#include <stdio.h>
#include "../../src/sp_config.h"
#include "../../src/capi/sp_cuda_common.h"
#include "../../src/capi/spMesh.h"
#include "../../src/capi/spField.h"
#include "../../src/capi/spParticle.h"
#include "Boris.h"
#include "BorisYee.h"

int main(int argc, char **argv)
{
	Real dt = 1.0;

	spMesh t_m;
	t_m.dims[0] = 10;
	t_m.dims[1] = 10;
	t_m.dims[2] = 10;
	t_m.dx[0] = 10;
	t_m.dx[1] = 10;
	t_m.dx[2] = 10;

#if defined(__CUDA_ARCH__)
	CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
#endif

	spMesh *mesh = 0x0;
	sp_particle_type *pg = 0x0;
	sp_field_type *d_fE = 0x0;
	sp_field_type *d_fB = 0x0;
	sp_field_type *d_fRho = 0x0;
	sp_field_type *d_fJ = 0x0;

	spCreateMesh(&mesh);

	sp_memcpy(mesh, &t_m, sizeof(spMesh));
	spInitializeMesh(mesh);

	spCreateField(&t_m, &d_fE, 1);
	spCreateField(&t_m, &d_fB, 1);
	spCreateField(&t_m, &d_fJ, 1);
	spCreateField(&t_m, &d_fRho, 1);

	int NUMBER_OF_PIC = 256;
	spCreateParticle(&t_m, &pg, sizeof(struct boris_point_s), 1.0, 1.0);

//	spInitializeParticle_BorisYee(mesh, pg, NUMBER_OF_PIC);
	CUDA_CHECK(NUMBER_OF_PIC);

	int count = 10;

//	while (count > 0)
//	{
//		spUpdateParticle_BorisYee(mesh, pg, dt, d_fE, d_fB, d_fRho, d_fJ);
//
//		spUpdateField_Yee(mesh, dt, d_fRho, d_fJ, d_fE, d_fB);
////        spSyncParticle(mesh, pg, MPI_COMMON_GLOBAL);
////        spSyncField(mesh, d_fJ, MPI_COMMON_GLOBAL);
////        spSyncField(mesh, d_fRho, MPI_COMMON_GLOBAL);
////
////        spWriteField(mesh, d_fRho, "/checkpoint/rho", SP_RECORD);
////
////        spSyncField(mesh, d_fE, MPI_COMMON_GLOBAL);
////        spSyncField(mesh, d_fB, MPI_COMMON_GLOBAL);
//
//		--count;
//	}
//
////    spWriteField(mesh, d_fE, "/dump/rho", SP_NEW);
////    spWriteField(mesh, d_fB, "/dump/rho", SP_NEW);
////    spWriteField(mesh, d_fJ, "/dump/rho", SP_NEW);
////    spWriteField(mesh, d_fRho, "/dump/rho", SP_NEW);
////    spWriteParticle(mesh, pg, "/dump/H", SP_NEW);
//

	spDestroyField(mesh, &d_fE);
	spDestroyField(mesh, &d_fB);
	spDestroyField(mesh, &d_fJ);
	spDestroyField(mesh, &d_fRho);
	CUDA_CHECK(0);
	spDestroyParticle(mesh, &pg);
	CUDA_CHECK(0);

	spDestroyMesh(&mesh);
#if defined(__CUDA_ARCH__)
	CUDA_CHECK_RETURN(hipDeviceReset());
#endif
	CUDA_CHECK(0);
	return 0;
}
