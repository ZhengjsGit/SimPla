/*
 * BorisYeeMain.c
 *
 *  Created on: 2016年6月15日
 *      Author: salmon
 */

#include "../../src/sp_config.h"
#include "../../src/capi/sp_cuda_common.h"
#include "../../src/capi/spMesh.h"
#include "../../src/capi/spField.h"
#include "../../src/capi/spParticle.h"
#include "BorisYee.h"

int
main(int argc, char **argv)
{
    Real dt = 1.0;

    spMesh *mesh = 0x0;
    sp_particle_type *pg = 0x0;
    sp_field_type *d_fE = 0x0;
    sp_field_type *d_fB = 0x0;
    sp_field_type *d_fRho = 0x0;
    sp_field_type *d_fJ = 0x0;

#if defined(__CUDA_ARCH__)
    CUDA_CHECK_RETURN(hipDeviceSynchronize ()); // Wait for the GPU launched work to complete
    CUDA_CHECK_RETURN(hipGetLastError ());
#endif
    spCreateMesh(&mesh);

    spCreateField(mesh, &d_fE, 1);
    spCreateField(mesh, &d_fB, 1);
    spCreateField(mesh, &d_fJ, 1);
    spCreateField(mesh, &d_fRho, 1);

    int NUMBER_OF_PIC = 256;
    spCreateParticle(mesh, &pg, NUMBER_OF_PIC);
    spInitializeParticle_BorisYee(mesh, pg);

    int count = 10;

    while (count > 0)
    {
        spUpdateParticle_BorisYee(mesh, pg, dt, d_fE, d_fB, d_fRho, d_fJ);

        spUpdateField_Yee(mesh, dt, d_fRho, d_fJ, d_fE, d_fB);
        spSyncParticle(mesh, pg, MPI_COMMON_GLOBAL);
        spSyncField(mesh, d_fJ, MPI_COMMON_GLOBAL);
        spSyncField(mesh, d_fRho, MPI_COMMON_GLOBAL);

        spWriteField(mesh, d_fRho, "/checkpoint/rho", SP_RECORD);

        spSyncField(mesh, d_fE, MPI_COMMON_GLOBAL);
        spSyncField(mesh, d_fB, MPI_COMMON_GLOBAL);


        --count;
    }

    spWriteField(mesh, d_fE, "/dump/rho", SP_NEW);
    spWriteField(mesh, d_fB, "/dump/rho", SP_NEW);
    spWriteField(mesh, d_fJ, "/dump/rho", SP_NEW);
    spWriteField(mesh, d_fRho, "/dump/rho", SP_NEW);
    spWriteParticle(mesh, pg, "/dump/H", SP_NEW);

    spDestroyField(mesh, &d_fE);
    spDestroyField(mesh, &d_fB);
    spDestroyField(mesh, &d_fJ);
    spDestroyField(mesh, &d_fRho);
    spDestroyParticle(mesh, &pg);
    spDestroyMesh(&mesh);
#if defined(__CUDA_ARCH__)
    CUDA_CHECK_RETURN(hipDeviceReset ());
#endif
    return 0;
}
