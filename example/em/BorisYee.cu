#include "hip/hip_runtime.h"
//
// Created by salmon on 16-6-14.
//

#include "BorisYee.h"
#include "Boris.h"

#include "../../src/sp_config.h"
#include "../../src/particle/BucketContainer.h"
//#include "../../src/particle/ParticleUtility.h

#include "../../src/particle/BucketContainer.c"

#define CACHE_EXTENT_X 4
#define CACHE_EXTENT_Y 4
#define CACHE_EXTENT_Z 4
#define CACHE_SIZE (CACHE_EXTENT_X*CACHE_EXTENT_Y*CACHE_EXTENT_Z)

#define IX  1
#define IY  CACHE_EXTENT_X
#define IZ  CACHE_EXTENT_X*CACHE_EXTENT_Y

#define ll 0
#define rr 1.0

__device__  void cache_gather(Real *v, Real const f[CACHE_SIZE], Real const *r0,
		const Real *r1)
{
	Real r[3] =
	{ r0[0] - r1[0], r0[1] - r1[1], r0[2] - r1[2] };
	id_type s = (int) (r[0]) * IX + (int) (r[1]) * IY + (int) (r[2]) * IZ;

	*v = f[s + IX + IY + IZ /*  */] * (r[0] - ll) * (r[1] - ll) * (r[2] - ll)
			+ f[s + IX + IY /*     */] * (r[0] - ll) * (r[1] - ll) * (rr - r[2])
			+ f[s + IX + IZ /*     */] * (r[0] - ll) * (rr - r[1]) * (r[2] - ll)
			+ f[s + IX /*          */] * (r[0] - ll) * (rr - r[1]) * (rr - r[2])
			+ f[s + IY + IZ /*     */] * (rr - r[0]) * (r[1] - ll) * (r[2] - ll)
			+ f[s + IY /*          */] * (rr - r[0]) * (r[1] - ll) * (rr - r[2])
			+ f[s + IZ /*          */] * (rr - r[0]) * (rr - r[1]) * (r[2] - ll)
			+ f[s /*               */] * (rr - r[0]) * (rr - r[1])
					* (rr - r[2]);
}

__device__ void cache_scatter(Real f[CACHE_SIZE], Real v, Real const *r0,
		Real const *r1)
{
	Real r[3] =
	{ r0[0] - r1[0], r0[1] - r1[1], r0[2] - r1[2] };
	id_type s = (int) (r[0]) * IX + (int) (r[1]) * IY + (int) (r[2]) * IZ;

	f[s + IX + IY + IZ /*  */] += v * (r[0] - ll) * (r[1] - ll) * (r[2] - ll);
	f[s + IX + IY /*       */] += v * (r[0] - ll) * (r[1] - ll) * (rr - r[2]);
	f[s + IX + IZ /*       */] += v * (r[0] - ll) * (rr - r[1]) * (r[2] - ll);
	f[s + IX /*            */] += v * (r[0] - ll) * (rr - r[1]) * (rr - r[2]);
	f[s + IY + IZ /*       */] += v * (rr - r[0]) * (r[1] - ll) * (r[2] - ll);
	f[s + IY /*            */] += v * (rr - r[0]) * (r[1] - ll) * (rr - r[2]);
	f[s + IZ /*            */] += v * (rr - r[0]) * (rr - r[1]) * (r[2] - ll);
	f[s/*                  */] += v * (rr - r[0]) * (rr - r[1]) * (rr - r[2]);

}

#undef ll
#undef rr
#undef IX
#undef IY
#undef IZ

/**
 *\verbatim
 *                ^y
 *               /
 *        z     /
 *        ^    /
 *    PIXEL0 110-------------111 VOXEL
 *        |  /|              /|
 *        | / |             / |
 *        |/  |    PIXEL1  /  |
 * EDGE2 100--|----------101  |
 *        | m |           |   |
 *        |  010----------|--011 PIXEL2
 *        |  / EDGE1      |  /
 *        | /             | /
 *        |/              |/
 *       000-------------001---> x
 *                       EDGE0
 *
 *\endverbatim
 */

/* @formatter:off*/
#define _R 1.0
__device__ Real id_to_shift_[][3] =
{ //
		{ 0, 0, 0 },           // 000
				{ _R, 0, 0 },           // 001
				{ 0, _R, 0 },           // 010
				{ 0, 0, _R },          // 011
				{ _R, _R, 0 },           // 100
				{ _R, 0, _R },          // 101
				{ 0, _R, _R },          // 110
				{ 0, _R, _R },          // 111
		};
__device__ int sub_index_to_id_[4][3] =
{ //
		{ 0, 0, 0 }, /*VERTEX*/
		{ 1, 2, 4 }, /*EDGE*/
		{ 6, 5, 3 }, /*FACE*/
		{ 7, 7, 7 } /*VOLUME*/

		};

#undef _R

/* @formatter:on*/
__device__
void spBorisYeeUpdate_kernel(BorisYeeUpdateArgs const *args, Real dt,
		spPage **first, spPage **second, spPagePool *pool, const Real *fE,
		const Real *fB, Real *fRho, Real *fJ)
{
	size_type entity_size_in_byte = (*first)->entity_size_in_byte;

	__shared__ Real tE[3][CACHE_SIZE], tB[3][CACHE_SIZE], tJ[4][CACHE_SIZE];
	__shared__ spPage *read_buffer[CACHE_SIZE];
//	__shared__ spPage *write_buffer[CACHE_SIZE];
//	__shared__ bucket_page_status_flag_t shift_flag[CACHE_SIZE];

	for (size_type _blk_s = blockIdx.x, _blk_e = args->number_of_idx;
			_blk_s < _blk_e; _blk_s += blockDim.x)
	{
		size_type cell_idx = args->cell_idx[_blk_s];

		// read tE,tB from E,B
		// clear tJ
		spPage **src = &first[cell_idx];
		spPage **dest = &second[cell_idx];
		id_type tag = 0x0;
		size_type num = spEntityCountIf(*src, tag);

		__shared__ byte_type write_buffer[sizeof(boris_point_s)
				* SP_NUMBER_OF_ENTITIES_IN_PAGE];
//		write_buffer=malloc( num * SP_NUMBER_OF_ENTITIES_IN_PAGE);

		// TODO load data to cache

		for (int n = 0; n < CACHE_SIZE; ++n)
		{

			spPage *pg = read_buffer[n];

			bucket_page_status_flag_t dest_flag = 0x0;

			bucket_page_status_flag_t read_flag = 0x0;

			bucket_page_status_flag_t write_flag = 0x0;

			for (boris_point_s *p0, *p1 = 0x0; (p0 =
					(boris_point_s *) spEntityNext(&pg, &read_flag)) != 0x0;)
			{
				if ((p0->_tag & 0x3F) == tag)
				{
					if (write_flag == 0x0 || write_buffer == 0x0)
					{
						/** atomic */

						/** atomic */
						spPage *t = spPageCreate(1, pool);

//						hipMemcpy(t->data, write_buffer,
//						SP_NUMBER_OF_ENTITIES_IN_PAGE * entity_size_in_byte,
//								hipMemcpyDeviceToDevice);

						t->flag = dest_flag;
						spPagePushFront(dest, t);
						dest_flag = 0x0;
						write_flag = 0x0;
						p1 = (boris_point_s *) write_buffer;
					}
					/****************************************************************/

					Real E[3], B[3];

					cache_gather(&E[0], tE[0], p0->r,
							id_to_shift_[sub_index_to_id_[1/*EDGE*/][0]]);
					cache_gather(&E[1], tE[1], p0->r,
							id_to_shift_[sub_index_to_id_[1/*EDGE*/][1]]);
					cache_gather(&E[2], tE[2], p0->r,
							id_to_shift_[sub_index_to_id_[1/*EDGE*/][2]]);

					cache_gather(&B[0], tB[0], p0->r,
							id_to_shift_[sub_index_to_id_[2/*FACE*/][0]]);
					cache_gather(&B[1], tB[1], p0->r,
							id_to_shift_[sub_index_to_id_[2/*FACE*/][1]]);
					cache_gather(&B[2], tB[2], p0->r,
							id_to_shift_[sub_index_to_id_[2/*FACE*/][2]]);

					spBorisPushOne(p0, p1, args->cmr, dt, E, B, args->inv_dx);

					cache_scatter(tJ[0], spBorisGetRho(p1), p1->r,
							id_to_shift_[sub_index_to_id_[0/*VERTEX*/][0]]);
					cache_scatter(tJ[1], spBorisGetJ(p1, 0), p1->r,
							id_to_shift_[sub_index_to_id_[1/*EDGE*/][0]]);
					cache_scatter(tJ[2], spBorisGetJ(p1, 1), p1->r,
							id_to_shift_[sub_index_to_id_[1/*EDGE*/][1]]);
					cache_scatter(tJ[3], spBorisGetJ(p1, 2), p1->r,
							id_to_shift_[sub_index_to_id_[1/*EDGE*/][2]]);

					/****************************************************************/
					p1->_tag &= ~(0x3F); // clear tag
					dest_flag |= write_flag;
					write_flag <<= 1;
					p1 += entity_size_in_byte;

				}
			}
			__syncthreads();

			//TODO atomic_add tJ to fJ
			for (int s = 0; s < CACHE_SIZE; ++s)
			{
//				size_type idx = posFromCacheIdx(s, args->i_dims);
//				atomicAdd(&(fRho[idx]), tJ[0][idx]);
//				atomicAdd(&(fJ[idx * 3 + 0]), tJ[1][idx]);
//				atomicAdd(&(fJ[idx * 3 + 0]), tJ[2][idx]);
//				atomicAdd(&(fJ[idx * 3 + 0]), tJ[3][idx]);
			}

		}        //foreach block

	}
}

void spBorisYeeUpdate(BorisYeeUpdateArgs const *args, Real dt, spPage **first,
		spPage **second, spPagePool *pool, const Real *fE, const Real *fB,
		Real *fRho, Real *fJ)
{
	//number_of_core / SP_NUMBER_OF_ELEMENT_IN_PAGE
//	int numBlocks = (16);
//
//	dim3 threadsPerBlock(SP_NUMBER_OF_ENTITIES_IN_PAGE, 1);
//
//	spBorisYeeUpdate_kernel<<<numBlocks, threadsPerBlock>>>(args, dt, first,
//			second, pool, fE, fB, fRho, fJ);

}
