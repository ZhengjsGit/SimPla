/*
 * BorisYeeMain.c
 *
 *  Created on: 2016年6月15日
 *      Author: salmon
 */

#include <stdio.h>
#include "../../sp_lite/sp_def.h"
#include "../../sp_lite/spMesh.h"
#include "../../sp_lite/spField.h"
#include "../../sp_lite/spParticle.h"
#include "Boris.h"
#include "BorisYee.h"

int main(int argc, char **argv)
{
	CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	spMesh *mesh;
	sp_particle_type *ps = 0x0;
	sp_field_type *fE = 0x0;
	sp_field_type *fB = 0x0;
	sp_field_type *fRho = 0x0;
	sp_field_type *fJ = 0x0;

	spCreateMesh(&mesh);
	mesh->dims[0] = 16;
	mesh->dims[1] = 16;
	mesh->dims[2] = 16;
	mesh->dx[0] = 1;
	mesh->dx[1] = 1;
	mesh->dx[2] = 1;
	spInitializeMesh(mesh);
	spCreateField(mesh, &fE, 1);
	spCreateField(mesh, &fB, 2);
	spCreateField(mesh, &fJ, 1);
	spCreateField(mesh, &fRho, 0);

	spClearField(mesh, fE);
	spClearField(mesh, fB);
	spClearField(mesh, fJ);
	spClearField(mesh, fRho);

	int NUMBER_OF_PIC = 256;
	spCreateParticle(mesh, &ps, sizeof(struct boris_point_s), NUMBER_OF_PIC);
	spInitializeParticle_BorisYee(mesh, ps, NUMBER_OF_PIC);

	int count = 1;
	Real dt = 1.0;

	spWriteField(mesh, fE, "/start/E", SP_NEW);
	spWriteField(mesh, fB, "/start/B", SP_NEW);
	spWriteField(mesh, fJ, "/start/J", SP_NEW);
	spWriteField(mesh, fRho, "/start/rho", SP_NEW);

	while (count > 0)
	{
		printf("====== REMINED STEP= %d ======\n", count);
		spUpdateParticle_BorisYee(mesh, dt, ps, fE, fB, fRho, fJ);
//		spUpdateField_Yee(mesh, dt, fRho, fJ, fE, fB);

//		spWriteField(mesh, fE, "/checkpoint/E", SP_RECORD);
//		spWriteField(mesh, fB, "/checkpoint/B", SP_RECORD);
//		spWriteField(mesh, fJ, "/checkpoint/J", SP_RECORD);
//		spWriteField(mesh, fRho, "/checkpoint/rho", SP_RECORD);

		--count;
	}
	printf("======  The End ======\n", count);

	spWriteField(mesh, fE, "/dump/E", SP_NEW);
	spWriteField(mesh, fB, "/dump/B", SP_NEW);
	spWriteField(mesh, fJ, "/dump/J", SP_NEW);
	spWriteField(mesh, fRho, "/dump/rho", SP_NEW);
//	spWriteParticle(mesh, pg, "/dump/H", SP_NEW);
	spDestroyField(&fE);
	spDestroyField(&fB);
	spDestroyField(&fJ);
	spDestroyField(&fRho);
	spDestroyParticle(&ps);
	spDestroyMesh(&mesh);

	CUDA_CHECK_RETURN(hipDeviceReset());
	DONE
	return 0;
}
